#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>

#include "hip/hip_runtime_api.h"
#include "NONLINEAR_SLICE_CUDA.h"
#include "TIMER.h"

// stop g++ from choking when Thrust is included
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <nppi.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>

__constant__ double4 topRoots[MAX_COEFFS];
__constant__ double4 bottomRoots[MAX_COEFFS];
__constant__ double topPowers[MAX_COEFFS];
__constant__ double bottomPowers[MAX_COEFFS];

__constant__ unsigned int totalTopRoots;
__constant__ unsigned int totalBottomRoots;

__constant__ int xRes;
__constant__ int yRes;
__constant__ int zRes;
__constant__ Real xLength;
__constant__ Real yLength;
__constant__ Real zLength;
__constant__ Real dx;
__constant__ Real dy;
__constant__ Real dz;
__constant__ Real xCenter;
__constant__ Real yCenter;
__constant__ Real zCenter;
__constant__ Real escape;
__constant__ Real expScaling;
__constant__ Real quaternionSlice;
__constant__ Real isosurface;
__constant__ int maxIterations;
__constant__ int binaryBandwidth;
__constant__ int curvatureBandwidth;

__device__ unsigned int blockCounter;   // global counter, initialized to zero before kernel launch

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
void setConsts(const int xResHost, const int yResHost, const int zResHost,
               const Real dxHost, const Real dyHost, const Real dzHost,
               const Real xCenterHost, const Real yCenterHost, const Real zCenterHost,
               const Real xLengthHost, const Real yLengthHost, const Real zLengthHost,
               const Real escapeHost, const int maxIterationsHost, const Real expScalingHost,
               const Real quaternionSliceHost, const Real isosurfaceHost)
{
  hipMemcpyToSymbol(HIP_SYMBOL(xRes), &xResHost, sizeof(int));
  getLastCudaError("xRes copy failed");
  hipMemcpyToSymbol(HIP_SYMBOL(yRes), &yResHost, sizeof(int));
  getLastCudaError("yRes copy failed");
  hipMemcpyToSymbol(HIP_SYMBOL(zRes), &zResHost, sizeof(int));
  getLastCudaError("zRes copy failed");

  hipMemcpyToSymbol(HIP_SYMBOL(dx), &dxHost, sizeof(Real));
  getLastCudaError("dx copy failed");
  hipMemcpyToSymbol(HIP_SYMBOL(dy), &dyHost, sizeof(Real));
  getLastCudaError("dy copy failed");
  hipMemcpyToSymbol(HIP_SYMBOL(dz), &dzHost, sizeof(Real));
  getLastCudaError("dz copy failed");

  hipMemcpyToSymbol(HIP_SYMBOL(xCenter), &xCenterHost, sizeof(Real));
  getLastCudaError("xCenter copy failed");
  hipMemcpyToSymbol(HIP_SYMBOL(yCenter), &yCenterHost, sizeof(Real));
  getLastCudaError("yCenter copy failed");
  hipMemcpyToSymbol(HIP_SYMBOL(zCenter), &zCenterHost, sizeof(Real));
  getLastCudaError("zCenter copy failed");

  hipMemcpyToSymbol(HIP_SYMBOL(xLength), &xLengthHost, sizeof(Real));
  getLastCudaError("xLength copy failed");
  hipMemcpyToSymbol(HIP_SYMBOL(yLength), &yLengthHost, sizeof(Real));
  getLastCudaError("yLength copy failed");
  hipMemcpyToSymbol(HIP_SYMBOL(zLength), &zLengthHost, sizeof(Real));
  getLastCudaError("zLength copy failed");

  hipMemcpyToSymbol(HIP_SYMBOL(escape), &escapeHost, sizeof(Real));
  getLastCudaError("escape copy failed");

  hipMemcpyToSymbol(HIP_SYMBOL(maxIterations), &maxIterationsHost, sizeof(int));
  getLastCudaError("maxIterations copy failed");
  
  hipMemcpyToSymbol(HIP_SYMBOL(expScaling), &expScalingHost, sizeof(Real));
  getLastCudaError("expScaling copy failed");
  
  hipMemcpyToSymbol(HIP_SYMBOL(quaternionSlice), &quaternionSliceHost, sizeof(Real));
  getLastCudaError("quaternionSlice copy failed");
  
  hipMemcpyToSymbol(HIP_SYMBOL(isosurface), &isosurfaceHost, sizeof(Real));
  getLastCudaError("isosurface copy failed");
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
hipError_t setTopRoots(double4* roots, unsigned int total)
{
  assert(total < MAX_COEFFS);
  hipMemcpyToSymbol(HIP_SYMBOL(totalTopRoots), &total,
                     sizeof(unsigned int));

  return hipMemcpyToSymbol(HIP_SYMBOL(topRoots), roots,
                            sizeof(double4) * total);
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
hipError_t setTopPowers(double* powers, unsigned int total)
{
  assert(total < MAX_COEFFS);

  return hipMemcpyToSymbol(HIP_SYMBOL(topPowers), powers,
                            sizeof(double) * total);
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
hipError_t setBottomRoots(double4* roots, unsigned int total)
{
  assert(total < MAX_COEFFS);
  hipMemcpyToSymbol(HIP_SYMBOL(totalBottomRoots), &total,
                     sizeof(unsigned int));

  return hipMemcpyToSymbol(HIP_SYMBOL(bottomRoots), roots,
                            sizeof(double4) * total);
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
hipError_t setBottomPowers(double* powers, unsigned int total)
{
  assert(total < MAX_COEFFS);

  return hipMemcpyToSymbol(HIP_SYMBOL(bottomPowers), powers,
                            sizeof(double) * total);
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline double4 make_quaternion(const double& w,
                                          const double& x,
                                          const double& y,
                                          const double& z)
{
  return make_double4(x,y,z,w);
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline double magnitudeSq4(const double4& data)
{
  return data.x * data.x + data.y * data.y + data.z * data.z + data.w * data.w;
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline double magnitude4(const double4& data)
{
  return sqrt(data.x * data.x + data.y * data.y + data.z * data.z + data.w * data.w);
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline double4 conjugateScaled(const double4& data, const double& scale)
{
  return make_quaternion(scale * data.w, 
                        -scale * data.x, 
                        -scale * data.y, 
                        -scale * data.z);
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline double4 conjugate(const double4& data)
{
  return make_quaternion(data.w, -data.x, -data.y, -data.z);
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline void multiply(const double4& left, const double4& right, double4& result)
{
  result.x = left.y * right.z - left.z * right.y + right.w * left.x + left.w * right.x;
  result.y = left.z * right.x - left.x * right.z + right.w * left.y + left.w * right.y;
  result.z = left.x * right.y - left.y * right.x + right.w * left.z + left.w * right.z;
  result.w = left.w * right.w - left.x * right.x - right.y * left.y - left.z * right.z;
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline double3 add(const double3& left, const double3& right)
{
  double3 result;
  result.x = left.x + right.x;
  result.y = left.y + right.y;
  result.z = left.z + right.z;
  return result;
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline void subtract(const double4& left, const double4& right, double4& result)
{
  result.x = left.x - right.x;
  result.y = left.y - right.y;
  result.z = left.z - right.z;
  result.w = left.w - right.w;
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline void scale(double4& left, const double& factor)
{
  left.x *= factor;
  left.y *= factor;
  left.z *= factor;
  left.w *= factor;
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline void scale(double3& left, const double& factor)
{
  left.x *= factor;
  left.y *= factor;
  left.z *= factor;
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ double4 pow(const double4& q, const double& exponent)
{
  const double partial = q.x * q.x + q.y * q.y + q.z * q.z;
  const double qMagnitude = sqrt(partial + q.w * q.w);
  const double vMagnitude = sqrt(partial);
  const double vMagnitudeInv = (vMagnitude > 0.0) ? 1.0 / vMagnitude : 0.0;

  const double scale = exponent * acos(q.w / qMagnitude) * vMagnitudeInv;

  const double magnitude = scale * vMagnitude;
  const double magnitudeInv = (magnitude > 0.0) ? 1.0 / magnitude : 0.0;

  const double exps = std::exp(exponent * std::log(qMagnitude));

  double sMag,cMag;
  sincos(magnitude, &sMag, &cMag);
  const double scale2 = scale * exps * magnitudeInv * sMag;
  return make_quaternion(exps * cMag, 
                         scale2 * q.x, 
                         scale2 * q.y, 
                         scale2 * q.z);
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline double4 evaluateBottom(const double4& iterate)
{
  double4 result, temp, temp2;

  result = make_quaternion(1.0, 0.0, 0.0, 0.0);

  for (int x = 0; x < totalBottomRoots; x++)
  {
    const double4& root = bottomRoots[x];
    temp.x = iterate.x - root.x;
    temp.y = iterate.y - root.y;
    temp.z = iterate.z - root.z;
    temp.w = iterate.w - root.w;
    
    // raise temp to a power
    temp2 = pow(temp, bottomPowers[x]);

    multiply(result, temp2, temp);
    result = temp;
  }

  return result;
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline double4 evaluateTop(const double4& iterate)
{
  double4 result, temp, temp2;
  subtract(iterate, topRoots[0], temp);

  // raise temp to a power
  result = pow(temp, topPowers[0]);

  for (int x = 1; x < totalTopRoots; x++)
  {
    const double4& root = topRoots[x];
    temp.x = iterate.x - root.x;
    temp.y = iterate.y - root.y;
    temp.z = iterate.z - root.z;
    temp.w = iterate.w - root.w;
    
    // raise temp to a power
    temp2 = pow(temp, topPowers[x]);

    multiply(result, temp2, temp);
    result = temp;
  }

  return result;
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline double3 cellCenter(const int x, const int y, const int z)
{
  return make_double3(xCenter - 0.5 * xLength + x * dx + dx * 0.5,
                      yCenter - 0.5 * yLength + y * dy + dy * 0.5,
                      zCenter - 0.5 * zLength + z * dz + dz * 0.5);
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
void __global__ computeCompactedFlags(const int* flags, 
                                      const int* indexTranslation, 
                                      int* compactedFlags)
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;

  // why doesn't this fix non-power-of-two problems?
  if (x >= xRes || y >= yRes) return;
  const int index = x + y * xRes;
  if (flags[index] == -1) return;

  compactedFlags[indexTranslation[index]] = flags[index];
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
void __global__ computeFlags(const double* slab0, 
                             const double* slab1, 
                             unsigned char* flags,
                             int* occupancies)
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;

  // why doesn't this fix non-power-of-two problems?
  //if (x >= xRes || y >= yRes) return;
  if (x >= xRes - 1 || y >= yRes - 1) return;

  const int index = x + y * xRes;
  //const double NNN = slab0(x,y);
  const double NNN = slab0[index];
  //const double NNP = slab1(x,y);
  const double NNP = slab1[index];
  //const double NPN = slab0(x,y + 1);
  const double NPN = slab0[index + xRes];
  //const double NPP = slab1(x,y + 1);
  const double NPP = slab1[index + xRes];
  //const double PNN = slab0(x + 1,y);
  const double PNN = slab0[index + 1];
  //const double PNP = slab1(x + 1,y);
  const double PNP = slab1[index + 1];
  //const double PPN = slab0(x + 1,y + 1);
  const double PPN = slab0[index + 1 + xRes];
  //const double PPP = slab1(x + 1,y + 1);
  const double PPP = slab1[index + 1 + xRes];
        
  unsigned char flag =    ((NNN > 0) + 2 *   (NNP > 0) + 4  * (NPN > 0) +
                       8 * (NPP > 0) + 16 *  (PNN > 0) + 32 * (PNP > 0) +
                       64 *(PPN > 0) + 128 * (PPP > 0));

  flags[index] = flag;

  int occupied = (flag == 0 || flag == 255) ? 0 : 1;
  occupancies[index] = occupied;
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
void __global__ computeFlags(const double* slab0, const double* slab1, unsigned char* flags)
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;

  // why doesn't this fix non-power-of-two problems?
  //if (x >= xRes || y >= yRes) return;
  if (x >= xRes - 1 || y >= yRes - 1) return;

  const int index = x + y * xRes;
  //const double NNN = slab0(x,y);
  const double NNN = slab0[index];
  //const double NNP = slab1(x,y);
  const double NNP = slab1[index];
  //const double NPN = slab0(x,y + 1);
  const double NPN = slab0[index + xRes];
  //const double NPP = slab1(x,y + 1);
  const double NPP = slab1[index + xRes];
  //const double PNN = slab0(x + 1,y);
  const double PNN = slab0[index + 1];
  //const double PNP = slab1(x + 1,y);
  const double PNP = slab1[index + 1];
  //const double PPN = slab0(x + 1,y + 1);
  const double PPN = slab0[index + 1 + xRes];
  //const double PPP = slab1(x + 1,y + 1);
  const double PPP = slab1[index + 1 + xRes];
        
  unsigned char flag =    ((NNN > 0) + 2 *   (NNP > 0) + 4  * (NPN > 0) +
                       8 * (NPP > 0) + 16 *  (PNN > 0) + 32 * (PNP > 0) +
                       64 *(PPN > 0) + 128 * (PPP > 0));

  flags[index] = flag;
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
__device__ inline double nonlinearValue(const double3& center)
{
  double4 iterate = make_quaternion(center.x, center.y, center.z, 
                                    quaternionSlice);
  double magnitude = magnitude4(iterate);
  int totalIterations = 0;

  double bottomPowerSum = bottomPowers[0];
  for (int x = 1; x < totalBottomRoots; x++)
    bottomPowerSum += bottomPowers[x];

  const double4 bail = make_double4(DBL_MAX, DBL_MAX, 
                                    DBL_MAX, DBL_MAX);
  bool bailed = false;
  while (magnitude < escape && totalIterations < maxIterations)
  {
    const double4 topEval = evaluateTop(iterate);
    const double4 bottomEval = evaluateBottom(iterate);
    const double bottomMagnitudeSq = magnitudeSq4(bottomEval);
    const double bottomMagnitude = sqrt(bottomMagnitudeSq);

    // compute guard value from Eqn. 2
    const double rhs = bottomPowerSum + 
                       log(bottomMagnitude) / log(10.0);
    const double topLimit = 308.0 - 1.05 * rhs;

    // if the division is tiny, bail
    if (bottomMagnitude < pow(10.0, topLimit))
    {
      const double4 bottomInv = 
        conjugateScaled(bottomEval, 1.0 / bottomMagnitudeSq);
      multiply(topEval, bottomInv, iterate);
    }
    else
    {
      iterate = bail;
      bailed = true;
    }

    const double scaling = (!bailed) ? expScaling : 1.0;
    scale(iterate, scaling);
    magnitude = (!bailed) ? magnitude4(iterate) : DBL_MAX;
    totalIterations++;
  }

  return log(magnitude);
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
void __global__ computeSlice(const int z, double* field)
{
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= xRes || y >= yRes) return;

  const int index = x + y * xRes;
  const double3 center = cellCenter(x,y,z);

  field[index] = nonlinearValue(center);
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
double3 __device__ midpointSearchForLoop(const double3& positiveVertex, 
                                         const double& positiveValue, 
                                         const double3& negativeVertex, 
                                         const double& negativeValue)
{
  double3 pVert = positiveVertex;
  double3 nVert = negativeVertex;

  double3 midpointVertex;
  double midpointValue;

  for (int x = 0; x <= 6; x++)
  {
      midpointVertex = add(pVert, nVert);
      scale(midpointVertex, 0.5);

      midpointValue = nonlinearValue(midpointVertex);
      if (fabs(midpointValue) < 1e-8) return midpointVertex;

      if (midpointValue < 0)
        nVert = midpointVertex;
      else
        pVert = midpointVertex;
  }
  return midpointVertex;
}

///////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////
void __global__ computeEdges(const int3* firstVertices, 
                             const int3* secondVertices,
                             const int size,
                             double3* finalVertices)
{
  const int index  = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= size) return;
    
  const int3& firstXYZ = firstVertices[index];
  const int3& secondXYZ = secondVertices[index];
  double3 firstVertex = cellCenter(firstXYZ.x, firstXYZ.y, firstXYZ.z);
  double3 secondVertex = cellCenter(secondXYZ.x, secondXYZ.y, secondXYZ.z);

  double firstValue = nonlinearValue(firstVertex);
  double secondValue = nonlinearValue(secondVertex);

  double3 positiveVertex = firstVertex;
  double positiveValue = firstValue;
  double3 negativeVertex = secondVertex;
  double negativeValue = secondValue;

  if (firstValue < 0)
  {
    positiveVertex = secondVertex;
    positiveValue = secondValue;
    negativeVertex = firstVertex;
    negativeValue = firstValue;
  }

  // this turns the midpoint search on and off. If you want to compare to just traditional
  // marching cubes with linear interpolation, set this to 0.
  double3 finalVertex = midpointSearchForLoop(positiveVertex, positiveValue, negativeVertex, negativeValue);
  finalVertices[index] = finalVertex;
}

#define BLOCK_SIZE 16
#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
void computeSliceOnGPU(const int xRes, 
                       const int yRes,
                       const int z, 
                       double* field)
{
    TIMER functionTimer(__FUNCTION__);
    dim3 gridSize((xRes + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (yRes + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y);
    dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    computeSlice<<<gridSize, blockSize>>>(z, field);

    hipDeviceSynchronize();
}

void computeFlagsOnGPU(const int xRes, 
                       const int yRes,
                       const double* slab0,
                       const double* slab1,
                       unsigned char* flags)
{
    dim3 gridSize((xRes + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (yRes + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y);
    dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    computeFlags<<<gridSize, blockSize>>>(slab0, slab1, flags);
}

void computeEdgesOnGPU(const int3* firstVertices,
                       const int3* secondVertices,
                       const int size,
                       double3* finalVertices)
{
    TIMER functionTimer(__FUNCTION__);
    if (size == 0) return;

    computeEdges<<<size / 64 + 1,64>>>(firstVertices, secondVertices, size, finalVertices);
    hipDeviceSynchronize();
}

int countFlagsOnGPU(const int xRes,
                    const int yRes,
                    const double* slab0,
                    const double* slab1,
                    unsigned char* flags,
                    int* occupancies,
                    int* indexTranslation,
                    int* compactedFlags)
{
    TIMER functionTimer(__FUNCTION__);
    dim3 gridSize((xRes + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (yRes + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y);
    dim3 blockSize(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    computeFlags<<<gridSize, blockSize>>>(slab0, slab1, flags, occupancies);

    TIMER reduceTimer("Thrust::reduce");
    thrust::device_ptr<int> oPtr(occupancies);
    int finalCount = thrust::reduce(oPtr, oPtr + xRes * yRes);
    reduceTimer.stop();

    return finalCount;
}
