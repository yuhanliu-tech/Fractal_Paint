#define GLM_FORCE_CUDA
#include "CudaMC.h"
#include <hip/hip_runtime.h>

// CUDA Constants and device variables
__constant__ int d_mc_internalMarching_cube_tris[256][16];
__device__ int defaultVerticeArraySize = 30000;
__device__ int defaultNormalArraySize = 30000;
__device__ int defaultTriangleArraySize = 60000;

void checkCUDAErrorFn(const char* msg, const char* file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}

void CudaMC::placeholder(int N) {
    int help = N;
}